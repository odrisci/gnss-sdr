#include "hip/hip_runtime.h"
/*!
 * \file cuda_multicorrelator.cu
 * \brief High optimized CUDA GPU vector multiTAP correlator class
 * \authors <ul>
 *          <li> Javier Arribas, 2015. jarribas(at)cttc.es
 *          </ul>
 *
 * Class that implements a high optimized vector multiTAP correlator class for NVIDIA CUDA GPUs
 *
 * -------------------------------------------------------------------------
 *
 * Copyright (C) 2010-2015  (see AUTHORS file for a list of contributors)
 *
 * GNSS-SDR is a software defined Global Navigation
 *          Satellite Systems receiver
 *
 * This file is part of GNSS-SDR.
 *
 * GNSS-SDR is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * GNSS-SDR is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with GNSS-SDR. If not, see <http://www.gnu.org/licenses/>.
 *
 * -------------------------------------------------------------------------
 */

///////////////////////////////////////////////////////////////////////////////
// On G80-class hardware 24-bit multiplication takes 4 clocks per warp
// (the same as for floating point  multiplication and addition),
// whereas full 32-bit multiplication takes 16 clocks per warp.
// So if integer multiplication operands are  guaranteed to fit into 24 bits
// (always lie withtin [-8M, 8M - 1] range in signed case),
// explicit 24-bit multiplication is preferred for performance.
///////////////////////////////////////////////////////////////////////////////
#define IMUL(a, b) __mul24(a, b)

#include "cuda_multicorrelator.h"

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>


#define ACCUM_N 256


__global__ void scalarProdGPUCPXxN_shifts_chips(
    GPU_Complex *d_corr_out,
    GPU_Complex *d_sig_in,
    GPU_Complex *d_local_code_in,
    float *d_shifts_chips,
    float code_length_chips,
    float code_phase_step_chips,
    float rem_code_phase_chips,
    int vectorN,
    int elementN
)
{
    //Accumulators cache
    __shared__ GPU_Complex accumResult[ACCUM_N];

    ////////////////////////////////////////////////////////////////////////////
    // Cycle through every pair of vectors,
    // taking into account that vector counts can be different
    // from total number of thread blocks
    ////////////////////////////////////////////////////////////////////////////
    for (int vec = blockIdx.x; vec < vectorN; vec += gridDim.x)
    {
        //int vectorBase = IMUL(elementN, vec);
        //int vectorEnd  = elementN;

        ////////////////////////////////////////////////////////////////////////
        // Each accumulator cycles through vectors with
        // stride equal to number of total number of accumulators ACCUM_N
        // At this stage ACCUM_N is only preferred be a multiple of warp size
        // to meet memory coalescing alignment constraints.
        ////////////////////////////////////////////////////////////////////////
        for (int iAccum = threadIdx.x; iAccum < ACCUM_N; iAccum += blockDim.x)
        {
        	GPU_Complex sum = GPU_Complex(0,0);

            for (int pos = iAccum; pos < elementN; pos += ACCUM_N)
            {
                //sum = sum + d_sig_in[pos-vectorBase] * d_nco_in[pos-vectorBase] * d_local_codes_in[pos];
            	//sum = sum + d_sig_in[pos-vectorBase] * d_local_codes_in[pos];
            	//sum.multiply_acc(d_sig_in[pos],d_local_codes_in[pos+d_shifts_samples[vec]]);

            	// 1.resample local code for the current shift
            	float local_code_chip_index= fmod(code_phase_step_chips*(float)pos + d_shifts_chips[vec] - rem_code_phase_chips, code_length_chips);
            	//TODO: Take into account that in multitap correlators, the shifts can be negative!
            	if (local_code_chip_index<0.0) local_code_chip_index+=code_length_chips;

            	// 2.correlate
            	sum.multiply_acc(d_sig_in[pos],d_local_code_in[__float2int_rd(local_code_chip_index)]);

            }
            accumResult[iAccum] = sum;
        }

        ////////////////////////////////////////////////////////////////////////
        // Perform tree-like reduction of accumulators' results.
        // ACCUM_N has to be power of two at this stage
        ////////////////////////////////////////////////////////////////////////
        for (int stride = ACCUM_N / 2; stride > 0; stride >>= 1)
        {
            __syncthreads();

            for (int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x)
            {
                accumResult[iAccum] += accumResult[stride + iAccum];
            }
        }

        if (threadIdx.x == 0)
        	{
        		d_corr_out[vec] = accumResult[0];
        	}
    }
}


///////////////////////////////////////////////////////////////////////////////
// Calculate scalar products of VectorN vectors of ElementN elements on GPU
// Parameters restrictions:
// 1) ElementN is strongly preferred to be a multiple of warp size to
//    meet alignment constraints of memory coalescing.
// 2) ACCUM_N must be a power of two.
///////////////////////////////////////////////////////////////////////////////


__global__ void scalarProdGPUCPXxN_shifts(
    GPU_Complex *d_corr_out,
    GPU_Complex *d_sig_in,
    GPU_Complex *d_local_codes_in,
    int *d_shifts_samples,
    int vectorN,
    int elementN
)
{
    //Accumulators cache
    __shared__ GPU_Complex accumResult[ACCUM_N];

    ////////////////////////////////////////////////////////////////////////////
    // Cycle through every pair of vectors,
    // taking into account that vector counts can be different
    // from total number of thread blocks
    ////////////////////////////////////////////////////////////////////////////
    for (int vec = blockIdx.x; vec < vectorN; vec += gridDim.x)
    {
        int vectorBase = IMUL(elementN, vec);
        int vectorEnd  = vectorBase + elementN;

        ////////////////////////////////////////////////////////////////////////
        // Each accumulator cycles through vectors with
        // stride equal to number of total number of accumulators ACCUM_N
        // At this stage ACCUM_N is only preferred be a multiple of warp size
        // to meet memory coalescing alignment constraints.
        ////////////////////////////////////////////////////////////////////////
        for (int iAccum = threadIdx.x; iAccum < ACCUM_N; iAccum += blockDim.x)
        {
        	GPU_Complex sum = GPU_Complex(0,0);

            for (int pos = vectorBase + iAccum; pos < vectorEnd; pos += ACCUM_N)
            {
                //sum = sum + d_sig_in[pos-vectorBase] * d_nco_in[pos-vectorBase] * d_local_codes_in[pos];
            	//sum = sum + d_sig_in[pos-vectorBase] * d_local_codes_in[pos];
            	sum.multiply_acc(d_sig_in[pos-vectorBase],d_local_codes_in[pos-vectorBase+d_shifts_samples[vec]]);
            }
            accumResult[iAccum] = sum;
        }

        ////////////////////////////////////////////////////////////////////////
        // Perform tree-like reduction of accumulators' results.
        // ACCUM_N has to be power of two at this stage
        ////////////////////////////////////////////////////////////////////////
        for (int stride = ACCUM_N / 2; stride > 0; stride >>= 1)
        {
            __syncthreads();

            for (int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x)
            {
                accumResult[iAccum] += accumResult[stride + iAccum];
            }
        }

        if (threadIdx.x == 0)
        	{
        		d_corr_out[vec] = accumResult[0];
        	}
    }
}


__global__ void scalarProdGPUCPXxN(
    GPU_Complex *d_corr_out,
    GPU_Complex *d_sig_in,
    GPU_Complex *d_local_codes_in,
    int vectorN,
    int elementN
)
{
    //Accumulators cache
    __shared__ GPU_Complex accumResult[ACCUM_N];

    ////////////////////////////////////////////////////////////////////////////
    // Cycle through every pair of vectors,
    // taking into account that vector counts can be different
    // from total number of thread blocks
    ////////////////////////////////////////////////////////////////////////////
    for (int vec = blockIdx.x; vec < vectorN; vec += gridDim.x)
    {
        //int vectorBase = IMUL(elementN, vec);
        //int vectorEnd  = vectorBase + elementN;

        ////////////////////////////////////////////////////////////////////////
        // Each accumulator cycles through vectors with
        // stride equal to number of total number of accumulators ACCUM_N
        // At this stage ACCUM_N is only preferred be a multiple of warp size
        // to meet memory coalescing alignment constraints.
        ////////////////////////////////////////////////////////////////////////
        for (int iAccum = threadIdx.x; iAccum < ACCUM_N; iAccum += blockDim.x)
        {
        	GPU_Complex sum = GPU_Complex(0,0);

            //for (int pos = vectorBase + iAccum; pos < vectorEnd; pos += ACCUM_N)
        	for (int pos = iAccum; pos < elementN; pos += ACCUM_N)
            {
                //sum = sum + d_sig_in[pos-vectorBase] * d_nco_in[pos-vectorBase] * d_local_codes_in[pos];
            	//sum = sum + d_sig_in[pos-vectorBase] * d_local_codes_in[pos];
            	//sum.multiply_acc(d_sig_in[pos-vectorBase],d_local_codes_in[pos]);
        		sum.multiply_acc(d_sig_in[pos],d_local_codes_in[pos]);
            }
            accumResult[iAccum] = sum;
        }

        ////////////////////////////////////////////////////////////////////////
        // Perform tree-like reduction of accumulators' results.
        // ACCUM_N has to be power of two at this stage
        ////////////////////////////////////////////////////////////////////////
        for (int stride = ACCUM_N / 2; stride > 0; stride >>= 1)
        {
            __syncthreads();

            for (int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x)
            {
                accumResult[iAccum] += accumResult[stride + iAccum];
            }
        }

        if (threadIdx.x == 0)
        	{
        		d_corr_out[vec] = accumResult[0];
        	}
    }
}


//*********** CUDA processing **************
// Treads: a minimal parallel execution code on GPU
// Blocks: a set of N threads
/**
 * CUDA Kernel Device code
 *
 * Computes the vectorial product of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void CUDA_32fc_x2_multiply_32fc(  GPU_Complex *A,   GPU_Complex  *B, GPU_Complex  *C, int numElements)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < numElements;
         i += blockDim.x * gridDim.x)
    {
        C[i] =  A[i] * B[i];
    }
}


/**
 * CUDA Kernel Device code
 *
 * Computes the carrier Doppler wipe-off by integrating the NCO in the CUDA kernel
 */
__global__ void
CUDA_32fc_Doppler_wipeoff(  GPU_Complex *sig_out, GPU_Complex *sig_in, float rem_carrier_phase_in_rad, float phase_step_rad, int numElements)
{
	//*** NCO CPU code (GNURadio FXP NCO)
	//float sin_f, cos_f;
	//float phase_step_rad = static_cast<float>(2 * GALILEO_PI) * d_carrier_doppler_hz / static_cast<float>(d_fs_in);
	//int phase_step_rad_i = gr::fxpt::float_to_fixed(phase_step_rad);
	//int phase_rad_i = gr::fxpt::float_to_fixed(d_rem_carr_phase_rad);
	//
	//for(int i = 0; i < d_current_prn_length_samples; i++)
	//    {
	//        gr::fxpt::sincos(phase_rad_i, &sin_f, &cos_f);
	//        d_carr_sign[i] = std::complex<float>(cos_f, -sin_f);
	//        phase_rad_i += phase_step_rad_i;
	//    }

	// CUDA version of floating point NCO and vector dot product integrated

    float sin;
    float cos;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < numElements;
         i += blockDim.x * gridDim.x)
    {
    	__sincosf(rem_carrier_phase_in_rad + i*phase_step_rad, &sin, &cos);
    	sig_out[i] =  sig_in[i] * GPU_Complex(cos,-sin);
    }
}


/**
 * CUDA Kernel Device code
 *
 * Computes the vectorial product of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void
CUDA_32fc_x2_add_32fc(  GPU_Complex *A,   GPU_Complex  *B, GPU_Complex  *C, int numElements)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < numElements;
         i += blockDim.x * gridDim.x)
    {
        C[i] =  A[i] + B[i];
    }
}


bool cuda_multicorrelator::init_cuda(const int argc, const char **argv, int signal_length_samples, int local_codes_length_samples, int n_correlators)
{
	// use command-line specified CUDA device, otherwise use device with highest Gflops/s
//	findCudaDevice(argc, (const char **)argv);
//      hipDeviceProp_t  prop;
//    int num_devices, device;
//    hipGetDeviceCount(&num_devices);
//    if (num_devices > 1) {
//          int max_multiprocessors = 0, max_device = 0;
//          for (device = 0; device < num_devices; device++) {
//                  hipDeviceProp_t properties;
//                  hipGetDeviceProperties(&properties, device);
//                  if (max_multiprocessors < properties.multiProcessorCount) {
//                          max_multiprocessors = properties.multiProcessorCount;
//                          max_device = device;
//                  }
//                  printf("Found GPU device # %i\n",device);
//          }
//          //hipSetDevice(max_device);
//
//          //set random device!
//          hipSetDevice(rand() % num_devices); //generates a random number between 0 and num_devices to split the threads between GPUs
//
//          hipGetDeviceProperties( &prop, max_device );
//          //debug code
//          if (prop.canMapHostMemory != 1) {
//              printf( "Device can not map memory.\n" );
//          }
//          printf("L2 Cache size= %u \n",prop.l2CacheSize);
//          printf("maxThreadsPerBlock= %u \n",prop.maxThreadsPerBlock);
//          printf("maxGridSize= %i \n",prop.maxGridSize[0]);
//          printf("sharedMemPerBlock= %lu \n",prop.sharedMemPerBlock);
//          printf("deviceOverlap= %i \n",prop.deviceOverlap);
//  	    printf("multiProcessorCount= %i \n",prop.multiProcessorCount);
//    }else{
//    	    int whichDevice;
//    	    hipGetDevice( &whichDevice );
//    	    hipGetDeviceProperties( &prop, whichDevice );
//    	    //debug code
//    	    if (prop.canMapHostMemory != 1) {
//    	        printf( "Device can not map memory.\n" );
//    	    }
//
//    	    printf("L2 Cache size= %u \n",prop.l2CacheSize);
//    	    printf("maxThreadsPerBlock= %u \n",prop.maxThreadsPerBlock);
//    	    printf("maxGridSize= %i \n",prop.maxGridSize[0]);
//    	    printf("sharedMemPerBlock= %lu \n",prop.sharedMemPerBlock);
//    	    printf("deviceOverlap= %i \n",prop.deviceOverlap);
//    	    printf("multiProcessorCount= %i \n",prop.multiProcessorCount);
//    }

	// (hipFuncSetCacheConfig(reinterpret_cast<const void*>(CUDA_32fc_x2_multiply_x2_dot_prod_32fc_), hipFuncCachePreferShared));


    // ALLOCATE GPU MEMORY FOR INPUT/OUTPUT and INTERNAL vectors

    size_t size = signal_length_samples * sizeof(GPU_Complex);

	hipMalloc((void **)&d_sig_in, size);
	// (hipMalloc((void **)&d_nco_in, size));
	hipMalloc((void **)&d_sig_doppler_wiped, size);

	// old version: all local codes are independent vectors
	// (hipMalloc((void **)&d_local_codes_in, size*n_correlators));

	// new version: only one vector with extra samples to shift the local code for the correlator set
	// Required: The last correlator tap in d_shifts_samples has the largest sample shift
    size_t size_local_code_bytes = local_codes_length_samples * sizeof(GPU_Complex);
	hipMalloc((void **)&d_local_codes_in, size_local_code_bytes);
	hipMalloc((void **)&d_shifts_samples, sizeof(int)*n_correlators);

	//scalars
	hipMalloc((void **)&d_corr_out, sizeof(std::complex<float>)*n_correlators);

    // Launch the Vector Add CUDA Kernel
	threadsPerBlock = 256;
    blocksPerGrid =(int)(signal_length_samples+threadsPerBlock-1)/threadsPerBlock;

	hipStreamCreate (&stream1) ;
	hipStreamCreate (&stream2) ;
	return true;
}


bool cuda_multicorrelator::init_cuda_integrated_resampler(
		const int argc, const char **argv,
		int signal_length_samples,
		int code_length_chips,
		int n_correlators
		)
{
	// use command-line specified CUDA device, otherwise use device with highest Gflops/s
//	findCudaDevice(argc, (const char **)argv);
//      hipDeviceProp_t  prop;
//    int num_devices, device;
//    hipGetDeviceCount(&num_devices);
//    if (num_devices > 1) {
//          int max_multiprocessors = 0, max_device = 0;
//          for (device = 0; device < num_devices; device++) {
//                  hipDeviceProp_t properties;
//                  hipGetDeviceProperties(&properties, device);
//                  if (max_multiprocessors < properties.multiProcessorCount) {
//                          max_multiprocessors = properties.multiProcessorCount;
//                          max_device = device;
//                  }
//                  printf("Found GPU device # %i\n",device);
//          }
//          //hipSetDevice(max_device);
//
//          //set random device!
//          hipSetDevice(rand() % num_devices); //generates a random number between 0 and num_devices to split the threads between GPUs
//
//          hipGetDeviceProperties( &prop, max_device );
//          //debug code
//          if (prop.canMapHostMemory != 1) {
//              printf( "Device can not map memory.\n" );
//          }
//          printf("L2 Cache size= %u \n",prop.l2CacheSize);
//          printf("maxThreadsPerBlock= %u \n",prop.maxThreadsPerBlock);
//          printf("maxGridSize= %i \n",prop.maxGridSize[0]);
//          printf("sharedMemPerBlock= %lu \n",prop.sharedMemPerBlock);
//          printf("deviceOverlap= %i \n",prop.deviceOverlap);
//  	    printf("multiProcessorCount= %i \n",prop.multiProcessorCount);
//    }else{
//    	    int whichDevice;
//    	    hipGetDevice( &whichDevice );
//    	    hipGetDeviceProperties( &prop, whichDevice );
//    	    //debug code
//    	    if (prop.canMapHostMemory != 1) {
//    	        printf( "Device can not map memory.\n" );
//    	    }
//
//    	    printf("L2 Cache size= %u \n",prop.l2CacheSize);
//    	    printf("maxThreadsPerBlock= %u \n",prop.maxThreadsPerBlock);
//    	    printf("maxGridSize= %i \n",prop.maxGridSize[0]);
//    	    printf("sharedMemPerBlock= %lu \n",prop.sharedMemPerBlock);
//    	    printf("deviceOverlap= %i \n",prop.deviceOverlap);
//    	    printf("multiProcessorCount= %i \n",prop.multiProcessorCount);
//    }

	// (hipFuncSetCacheConfig(reinterpret_cast<const void*>(CUDA_32fc_x2_multiply_x2_dot_prod_32fc_), hipFuncCachePreferShared));

    // ALLOCATE GPU MEMORY FOR INPUT/OUTPUT and INTERNAL vectors

    size_t size = signal_length_samples * sizeof(GPU_Complex);

	hipMalloc((void **)&d_sig_in, size);
	hipMemset(d_sig_in,0,size);

	// (hipMalloc((void **)&d_nco_in, size));
	hipMalloc((void **)&d_sig_doppler_wiped, size);
	hipMemset(d_sig_doppler_wiped,0,size);

	hipMalloc((void **)&d_local_codes_in, sizeof(std::complex<float>)*code_length_chips);
	hipMemset(d_local_codes_in,0,sizeof(std::complex<float>)*code_length_chips);

    d_code_length_chips=code_length_chips;

	hipMalloc((void **)&d_shifts_chips, sizeof(float)*n_correlators);
	hipMemset(d_shifts_chips,0,sizeof(float)*n_correlators);

	//scalars
	hipMalloc((void **)&d_corr_out, sizeof(std::complex<float>)*n_correlators);
	hipMemset(d_corr_out,0,sizeof(std::complex<float>)*n_correlators);

    // Launch the Vector Add CUDA Kernel
	threadsPerBlock = 256;
    blocksPerGrid =(int)(signal_length_samples+threadsPerBlock-1)/threadsPerBlock;

	hipStreamCreate (&stream1) ;
	hipStreamCreate (&stream2) ;
	return true;
}

bool cuda_multicorrelator::set_local_code_and_taps(
		int code_length_chips,
		const std::complex<float>* local_codes_in,
		float *shifts_chips,
		int n_correlators
		)
{
    // local code CPU -> GPU copy memory
    hipMemcpyAsync(d_local_codes_in, local_codes_in, sizeof(GPU_Complex)*code_length_chips, hipMemcpyHostToDevice,stream1);
    d_code_length_chips=(float)code_length_chips;

    // Correlator shifts vector CPU -> GPU copy memory (fractional chip shifts are allowed!)
    hipMemcpyAsync(d_shifts_chips, shifts_chips, sizeof(float)*n_correlators,
                                    hipMemcpyHostToDevice,stream1);

	return true;
}



bool cuda_multicorrelator::Carrier_wipeoff_multicorrelator_cuda(
		std::complex<float>* corr_out,
		const std::complex<float>* sig_in,
		const std::complex<float>* local_codes_in,
		float rem_carrier_phase_in_rad,
		float phase_step_rad,
		const int *shifts_samples,
		int signal_length_samples,
		int n_correlators)
	{

	size_t memSize = signal_length_samples * sizeof(std::complex<float>);

	// input signal CPU -> GPU copy memory

    hipMemcpyAsync(d_sig_in, sig_in, memSize,
                                    hipMemcpyHostToDevice, stream1);

    //***** NOTICE: NCO is computed on-the-fly, not need to copy NCO into GPU! ****
    // (hipMemcpyAsync(d_nco_in, nco_in, memSize,
    //                                hipMemcpyHostToDevice, stream1));


	// old version: all local codes are independent vectors
    // (hipMemcpyAsync(d_local_codes_in, local_codes_in, memSize*n_correlators,
    //                                hipMemcpyHostToDevice, stream2));

	// new version: only one vector with extra samples to shift the local code for the correlator set
	// Required: The last correlator tap in d_shifts_samples has the largest sample shift

    // local code CPU -> GPU copy memory
    hipMemcpyAsync(d_local_codes_in, local_codes_in, memSize+sizeof(std::complex<float>)*shifts_samples[n_correlators-1],
                                    hipMemcpyHostToDevice, stream2);
    // Correlator shifts vector CPU -> GPU copy memory
    hipMemcpyAsync(d_shifts_samples, shifts_samples, sizeof(int)*n_correlators,
                                    hipMemcpyHostToDevice, stream2);


    //Launch carrier wipe-off kernel here, while local codes are being copied to GPU!
    hipStreamSynchronize(stream1);
    CUDA_32fc_Doppler_wipeoff<<<blocksPerGrid, threadsPerBlock,0, stream1>>>(d_sig_doppler_wiped, d_sig_in,rem_carrier_phase_in_rad,phase_step_rad, signal_length_samples);


    //printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    //wait for Doppler wipeoff end...
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);
    // (hipDeviceSynchronize());

    //old
//    scalarProdGPUCPXxN<<<blocksPerGrid, threadsPerBlock,0 ,stream2>>>(
//    		d_corr_out,
//    		d_sig_doppler_wiped,
//    		d_local_codes_in,
//            3,
//            signal_length_samples
//        );

    //new
    //launch the multitap correlator
    scalarProdGPUCPXxN_shifts<<<blocksPerGrid, threadsPerBlock,0 ,stream2>>>(
			d_corr_out,
			d_sig_doppler_wiped,
			d_local_codes_in,
			d_shifts_samples,
			n_correlators,
			signal_length_samples
		);
    hipGetLastError();
    //wait for correlators end...
    hipStreamSynchronize(stream2);
    // Copy the device result vector in device memory to the host result vector
    // in host memory.

    //scalar products (correlators outputs)
    hipMemcpy(corr_out, d_corr_out, sizeof(std::complex<float>)*n_correlators,
            hipMemcpyDeviceToHost);
    return true;
}

bool cuda_multicorrelator::Carrier_wipeoff_multicorrelator_resampler_cuda(
		std::complex<float>* corr_out,
		const std::complex<float>* sig_in,
		float rem_carrier_phase_in_rad,
		float phase_step_rad,
        float code_phase_step_chips,
        float rem_code_phase_chips,
		int signal_length_samples,
		int n_correlators)
	{

	size_t memSize = signal_length_samples * sizeof(std::complex<float>);
	// input signal CPU -> GPU copy memory
    hipMemcpyAsync(d_sig_in, sig_in, memSize,
                                    hipMemcpyHostToDevice, stream2);

    //***** NOTICE: NCO is computed on-the-fly, not need to copy NCO into GPU! ****

    //Launch carrier wipe-off kernel here, while local codes are being copied to GPU!
    hipStreamSynchronize(stream2);

    CUDA_32fc_Doppler_wipeoff<<<blocksPerGrid, threadsPerBlock,0, stream2>>>(d_sig_doppler_wiped, d_sig_in,rem_carrier_phase_in_rad,phase_step_rad, signal_length_samples);

    //wait for Doppler wipeoff end...
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    //launch the multitap correlator with integrated local code resampler!

    scalarProdGPUCPXxN_shifts_chips<<<blocksPerGrid, threadsPerBlock,0 ,stream1>>>(
			d_corr_out,
			d_sig_doppler_wiped,
			d_local_codes_in,
			d_shifts_chips,
			d_code_length_chips,
	        code_phase_step_chips,
	        rem_code_phase_chips,
			n_correlators,
			signal_length_samples
		);

    hipGetLastError();
    //wait for correlators end...
    hipStreamSynchronize(stream1);
    // Copy the device result vector in device memory to the host result vector
    // in host memory.

    //scalar products (correlators outputs)
    hipMemcpyAsync(corr_out, d_corr_out, sizeof(std::complex<float>)*n_correlators,
            hipMemcpyDeviceToHost,stream1);
    hipStreamSynchronize(stream1);
    return true;
}


cuda_multicorrelator::cuda_multicorrelator()
{
	d_sig_in=NULL;
	d_nco_in=NULL;
	d_sig_doppler_wiped=NULL;
	d_local_codes_in=NULL;
	d_shifts_samples=NULL;
	d_shifts_chips=NULL;
	d_corr_out=NULL;
	threadsPerBlock=0;
	blocksPerGrid=0;
	d_code_length_chips=0;
}

bool cuda_multicorrelator::free_cuda()
{
	// Free device global memory
	if (d_sig_in!=NULL) hipFree(d_sig_in);
	if (d_nco_in!=NULL) hipFree(d_nco_in);
	if (d_sig_doppler_wiped!=NULL) hipFree(d_sig_doppler_wiped);
	if (d_local_codes_in!=NULL) hipFree(d_local_codes_in);
	if (d_corr_out!=NULL) hipFree(d_corr_out);


	if (d_shifts_samples!=NULL) hipFree(d_shifts_samples);
	if (d_shifts_chips!=NULL) hipFree(d_shifts_chips);


	hipStreamDestroy(stream1) ;
	hipStreamDestroy(stream2) ;

    // Reset the device and exit
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
	// (hipDeviceReset());
	return true;
}

